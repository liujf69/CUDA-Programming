#include "hip/hip_runtime.h"
#include "./common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// 随机化初始测试数据
void initialData(float *ip, const int size){
    int i;
    for(i = 0; i < size; i++){
        ip[i] = (float)( rand() & 0xFF ) / 10.0f;
    }
}

void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny){
    float *ia = A;
    float *ib = B;
    float *ic = C;

    for (int iy = 0; iy < ny; iy++){
        for (int ix = 0; ix < nx; ix++){
            ic[ix] = ia[ix] + ib[ix];
        }

        ia += nx;
        ib += nx;
        ic += nx;
    }
    return;
}

void checkResult(float *hostRef, float *gpuRef, const int N){
    double epsilon = 1.0E-8;

    for (int i = 0; i < N; i++){
        if (abs(hostRef[i] - gpuRef[i]) > epsilon){
            printf("host %f gpu %f ", hostRef[i], gpuRef[i]);
            printf("Arrays do not match.\n\n");
            break;
        }
    }
}

__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, int NX, int NY){
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * NX + ix;

    if (ix < NX && iy < NY){
        C[idx] = A[idx] + B[idx];
    }
}

int main(int argc, char **argv){
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    CHECK(hipSetDevice(dev));

    int nx = 1 << 14;
    int ny = 1 << 14;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    size_t iStart = seconds();
    initialData(h_A, nxy);
    initialData(h_B, nxy);
    size_t iElaps = seconds() - iStart;

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    iStart = seconds();
    sumMatrixOnHost (h_A, h_B, hostRef, nx, ny);
    iElaps = seconds() - iStart;

    float *d_MatA, *d_MatB, *d_MatC;
    CHECK(hipMalloc((void **)&d_MatA, nBytes));
    CHECK(hipMalloc((void **)&d_MatB, nBytes));
    CHECK(hipMalloc((void **)&d_MatC, nBytes));

    CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;

    if(argc > 2){ // 设置线程块大小
        dimx = atoi(argv[1]);
        dimy = atoi(argv[2]);
    }

    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    CHECK(hipDeviceSynchronize());
    iStart = seconds();
    sumMatrixOnGPU2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    // printf("sumMatrixOnGPU2D <<<(%d,%d), (%d,%d)>>> elapsed %d ms\n", grid.x,
    //        grid.y,
    //        block.x, block.y, iElaps);
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));

    checkResult(hostRef, gpuRef, nxy);

    CHECK(hipFree(d_MatA));
    CHECK(hipFree(d_MatB));
    CHECK(hipFree(d_MatC));

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
